
#include <hip/hip_runtime.h>
#include <stdio.h> 

// Used to verify that the data is serialized correctly device-side
#define TEST_VALUE -12345 

class ConstructorExample {
public:
	int var;
	
	ConstructorExample(int var) {
		this->var = var; 
	}
	
	// Since this class has a copy constructor, you cannot pass it 
	// to a device-side kernel launch.
	__host__ __device__ ConstructorExample(const ConstructorExample& other) {
		var = other.var; 
	}
};

#if defined (DEVICE_COPY_CONSTRUCTOR_ERROR)

// This is what we would normally do, but ConstructorExample has a copy
// constructor so this will yield an error. 
__global__ void kernel_sub(ConstructorExample obj) {
	printf("Sub-kernel invoked: %d\n", obj.var); 
}

#else 

// Instead, pass a byte array containing the data of the class we want
// to pass. 
__global__ void kernel_sub(char *obj_serialized) {
	ConstructorExample obj = *(static_cast<ConstructorExample*>((void*)obj_serialized));
	
	printf("Sub-kernel invoked\n");
	if (obj.var != TEST_VALUE) {
		printf("Error: passed object not serialized correctly (%d)\n", obj.var); 
	}
}

#endif 

__global__ void kernel_base(ConstructorExample obj) {
	printf("Base-kernel invoked\n"); 
	
#if defined(DEVICE_COPY_CONSTRUCTOR_ERROR)
	
	// Since obj has a copy constructor, this will yield an error. 
	kernel_sub<<<1,1>>>(obj); 

#else
	
	// Copy the data to a byte array and pass that instead. 
	char *serialized = (char*)malloc(sizeof(obj)); 
	memcpy(serialized, &obj, sizeof(obj)); 
	kernel_sub<<<1,1>>>(serialized); 
	free(serialized); 
	
#endif 

}

int main(void) {
	ConstructorExample obj1(TEST_VALUE); 
	ConstructorExample obj2 = obj1; // Copy constructor invoked
	
	if (obj1.var != obj2.var) {
		printf("Error: copy-constructor not being invoked (%d != %d)\n", obj1.var, obj2.var); 
		exit(1); 
	}
	
	printf("Starting kernel example...\n"); 
	kernel_base<<<1,1>>>(obj1);
	
	hipError_t launch_error = hipPeekAtLastError(); 
	hipError_t synchronize_error = hipDeviceSynchronize(); 
	if (launch_error != hipSuccess || synchronize_error != hipSuccess) {
		printf("Error in kernel: %s\n", hipGetErrorString(launch_error != hipSuccess ? launch_error : synchronize_error)); 
		exit(1); 
	}
	
	printf("Finished!\n"); 
}